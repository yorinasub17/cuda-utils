// Copyright 2014, Captricity Inc.
// All rights reserved.
//
// Author: yoriy@captricity.com
//
// CUDA Utilities
//
// Contains various utility functions that are callable from CPP

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"

void warmCuda()
{
    hipFree(0);
}

void waitForDevice()
{
    hipDeviceSynchronize();
}

void checkError(char* message)
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    }
}

void checkMemory()
{
    size_t free_byte, total_byte;
    double free_db, total_db, used_db;
    hipMemGetInfo(&free_byte, &total_byte);
    free_db = (double)free_byte;
    total_db = (double)total_byte;
    used_db = total_db - free_db;
    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}
